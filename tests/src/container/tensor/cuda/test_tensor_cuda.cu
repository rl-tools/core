#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#define RL_TOOLS_OPERATIONS_CPU_MUX_INCLUDE_CUDA
#include <rl_tools/operations/cpu_mux.h>

namespace rlt = rl_tools;


using DEVICE = rlt::devices::DEVICE_FACTORY_CUDA<>;
using DEVICE_CPU = rlt::devices::DEVICE_FACTORY<>;


template <typename T, typename TI, typename SHAPE, typename DEVICE, typename DEVICE_CPU>
void test(DEVICE& device, DEVICE_CPU& device_cpu, T epsilon) {
    epsilon *= pow(10, SHAPE::LENGTH);
    rlt::Tensor<rlt::tensor::Specification<T, TI, SHAPE, true>> tensor, tensor_cpu, target;
    rlt::malloc(device, tensor);
    rlt::malloc(device_cpu, tensor_cpu);
    rlt::malloc(device_cpu, target);
    {
        rlt::set_all(device, tensor, 1337);
        rlt::set_all(device_cpu, target, 1337);
        rlt::copy(device, device_cpu, tensor, tensor_cpu);
        T diff = rlt::abs_diff(device_cpu, tensor_cpu, target);
        std::cout << "set_all diff: " << diff << std::endl;
        ASSERT_EQ(diff, 0);
    }
    {
        rlt::set_all(device, tensor, 1.3);
        rlt::set_all(device_cpu, target, 1.3);
        rlt::exp(device, tensor);
        rlt::exp(device_cpu, target);
        rlt::copy(device, device_cpu, tensor, tensor_cpu);
        T diff = rlt::abs_diff(device_cpu, tensor_cpu, target);
        if constexpr(SHAPE::LENGTH == 1 && SHAPE::template GET<0> < 100) {
            rlt::print(device_cpu, target);
        }
        std::cout << "exp diff: " << diff << std::endl;
        ASSERT_LT(diff, epsilon);
    }
    {
        rlt::Tensor<rlt::tensor::Specification<T, TI, rlt::tensor::Shape<TI, 1>, true>> result, result_cpu, result_target;
        rlt::malloc(device, result);
        rlt::malloc(device_cpu, result_cpu);
        rlt::malloc(device_cpu, result_target);
        rlt::set_all(device, tensor, 1.3);
        rlt::set_all(device_cpu, tensor_cpu, 1.3);
        rlt::cast_reduce_sum<T>(device, tensor, result);
        rlt::cast_reduce_sum<T>(device_cpu, tensor_cpu, result_target);
        rlt::copy(device, device_cpu, result, result_cpu);
        T diff = rlt::abs_diff(device_cpu, result_cpu, result_target);
        std::cout << "Result target: " << rlt::get(device, result_target, 0) << std::endl;
        std::cout << "Result cpu: " << rlt::get(device_cpu, result_cpu, 0) << std::endl;
        std::cout << "unary associative reduce: sum diff: " << diff << std::endl;
        ASSERT_LT(diff, epsilon);
        rlt::free(device, result);
        rlt::free(device_cpu, result_cpu);
        rlt::free(device_cpu, result_target);
    }
}

template <typename T>
void test(T epsilon){
    DEVICE device;
    DEVICE_CPU device_cpu;
    rlt::init(device);
    using TI = typename DEVICE::index_t;
    test<T, TI, rlt::tensor::Shape<TI, 1>>(device, device_cpu, epsilon);
    test<T, TI, rlt::tensor::Shape<TI, 2>>(device, device_cpu, epsilon);
    test<T, TI, rlt::tensor::Shape<TI, 10>>(device, device_cpu, epsilon);
    test<T, TI, rlt::tensor::Shape<TI, 10, 1>>(device, device_cpu, epsilon);
    test<T, TI, rlt::tensor::Shape<TI, 1, 10>>(device, device_cpu, epsilon);
    test<T, TI, rlt::tensor::Shape<TI, 10, 10>>(device, device_cpu, epsilon);
    test<T, TI, rlt::tensor::Shape<TI, 10, 10, 1>>(device, device_cpu, epsilon);
    test<T, TI, rlt::tensor::Shape<TI, 10, 1, 10>>(device, device_cpu, epsilon);
    test<T, TI, rlt::tensor::Shape<TI, 1, 10, 10>>(device, device_cpu, epsilon);
    test<T, TI, rlt::tensor::Shape<TI, 10, 10, 10>>(device, device_cpu, epsilon);
}

TEST(RL_TOOLS_CONTAINER_TENSOR_CUDA, FP64){
    using T = double;
    T epsilon = 1e-13;
    test<T>(epsilon);
}

TEST(RL_TOOLS_CONTAINER_TENSOR_CUDA, FP32){
    using T = float;
    T epsilon = 1e-5;
    test<T>(epsilon);
}

